
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

int main()
{
    int n;
    cout << "Input the number of nodes: ";
    cin >> n;
    int m;
    // cout<<"Input the number of edges: ";
    // cin>>m;

    int adjacency[n + 1][n + 1];

    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            cout << "Input for " << i << " " << j << " index: ";
            cin >> adjacency[i][j];
        }
    }

    /* for(int i=0;i<n;i++){
        for(int j=0;j<n;j++){

            cout<<adjacency[i][j]<<" ";
        }
        cout<<endl;
    }

*/
    int inDegree = 0, outDegree = 0;
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            if (adjacency[i][j] == 1)
            {
                inDegree++;
            }
            if (adjacency[j][i] == 1)
            {
                outDegree++;
            }
        }
    }

    cout << "Indegree: " << inDegree << " OutDegree: " << outDegree << endl;

    cout << "Sum of in degree and out degree: " << inDegree + outDegree << endl;
    return 0;
}